#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>
#include <hip/hip_vector_types.h>
#include <algorithm>
#include <cmath>
#include <iostream>
#include <stdio.h>
#include "inline.cuh"
#include "cuda_interface.h"
#include "RayTracer.cuh"
#include "RayProcessing.cuh"

__constant__ RayTracingParams d_params;

// Host function to initialize the GPU with constant parameters
extern "C" hipError_t InitializeGPUKernel(const RayTracingParams* params)
{
    // Copy the parameters to the device's constant memory
    return hipMemcpyToSymbol((const void*)&d_params, (const void*)params, sizeof(RayTracingParams));

    void* d_addr;
    hipError_t error;

    // Get the address of the symbol in device memory
    error = hipGetSymbolAddress(&d_addr, HIP_SYMBOL((const void*))&d_params);
    if (error != hipSuccess) {
        return error;
    }

    // Copy the data to the symbol
    error = hipMemcpy(d_addr, params, sizeof(RayTracingParams), hipMemcpyHostToDevice);
    return error;
}

extern "C" hipError_t InitializeTransformMatrix(const float* positionMatrix)
{
    return hipMemcpyToSymbol(HIP_SYMBOL(cudaTrans), positionMatrix, sizeof(float) * DimTotal * (DimTotal + 1));
}

static int getGridX(int raysPerLine, int totalLines, int xBlocks) {

    const int threadsPerBlock = 256; // 32 * 8 * 1
    const int warpsPerBlock = threadsPerBlock / 32;
    const int blocksPerSM = 8; // This is a general value, might need tuning
    const int numSMs = 20; // For GTX 1080

    // Calculate total threads needed
    int totalThreadsNeeded = raysPerLine * totalLines;

    // Calculate optimal number of blocks
    int optimalNumBlocks = (totalThreadsNeeded + threadsPerBlock - 1) / threadsPerBlock;

    // Limit the number of blocks to a multiple of the optimal blocks per SM
    int limitedNumBlocks = std::min(optimalNumBlocks, blocksPerSM * numSMs);
    limitedNumBlocks = (limitedNumBlocks + warpsPerBlock - 1) / warpsPerBlock * warpsPerBlock;

    // Return grid X dimension
    return std::min(limitedNumBlocks, (int)((raysPerLine + xBlocks - 1) / xBlocks));
}

extern "C" hipError_t LaunchProcessRaysKernel(const RayTracingParams* rayParams, const RenderingParams* renderParams,
    int raysPerLine, int totalLines, ProgressCallback callback)
{
    // Allocate device memory
    RayDataTypeIntermediate* d_results;
    hipError_t cudaStatus = hipMalloc(&d_results, raysPerLine * totalLines * sizeof(RayDataTypeIntermediate));
    if (cudaStatus != hipSuccess) {
        return cudaStatus;
    }
 
    dim3 blockDim(32, 8, 1);
    dim3 gridDim(
        getGridX(raysPerLine, totalLines, blockDim.x),
        (totalLines + blockDim.y - 1) / blockDim.y);

    // Launch kernel
    ProcessRaysKernel << <gridDim, blockDim >> > (*rayParams, *renderParams, raysPerLine, totalLines, d_results);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {      
        hipFree(d_results);
        return cudaStatus;
    }

    // Allocate host memory
    RayDataTypeIntermediate* h_results = new RayDataTypeIntermediate[raysPerLine * totalLines];

    // Copy results back to host
    cudaStatus = hipMemcpy(h_results, d_results, raysPerLine * totalLines * sizeof(RayDataTypeIntermediate), hipMemcpyDeviceToHost);

    if (cudaStatus != hipSuccess) {
        std::cout << "Cuda error:" << hipGetErrorString(cudaStatus) << std::endl;
        delete[] h_results;
        hipFree(d_results);
        return cudaStatus;
    }

    // Process results and call the callback for each ray
    for (int rowCount = 0; rowCount < totalLines; ++rowCount) {
        for (int rayCount = 0; rayCount < raysPerLine; ++rayCount) {
            int index = rowCount * raysPerLine + rayCount;
            if (callback) {
                callback(rayCount, rowCount, &h_results[index]);
            }
        }
    }

    // Free memory
    delete[] h_results;
    hipFree(d_results);

    return hipSuccess;
}