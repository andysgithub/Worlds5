#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>
#include <hip/hip_vector_types.h>
#include <cmath>
#include <iostream>
#include <stdio.h>
#include "inline.cuh"
#include "cuda_interface.h"
#include "RayTracer.cuh"
#include "RayProcessing.cuh"

__constant__ RayTracingParams d_params;

// Host function to initialize the GPU with constant parameters
extern "C" hipError_t InitializeGPUKernel(const RayTracingParams* params)
{
    // Copy the parameters to the device's constant memory
    return hipMemcpyToSymbol((const void*)&d_params, (const void*)params, sizeof(RayTracingParams));

    void* d_addr;
    hipError_t error;

    // Get the address of the symbol in device memory
    error = hipGetSymbolAddress(&d_addr, HIP_SYMBOL((const void*))&d_params);
    if (error != hipSuccess) {
        return error;
    }

    // Copy the data to the symbol
    error = hipMemcpy(d_addr, params, sizeof(RayTracingParams), hipMemcpyHostToDevice);
    return error;
}

extern "C" hipError_t InitializeTransformMatrix(const float* positionMatrix)
{
    return hipMemcpyToSymbol(HIP_SYMBOL(cudaTrans), positionMatrix, sizeof(float) * DimTotal * (DimTotal + 1));
}

extern "C" hipError_t LaunchProcessRaysKernel(const RayTracingParams* rayParams, const RenderingParams* renderParams,
    int raysPerLine, int totalLines, ProgressCallback callback)
{
    // Allocate device memory
    RayDataTypeIntermediate* d_results;
    hipError_t cudaStatus = hipMalloc(&d_results, raysPerLine * totalLines * sizeof(RayDataTypeIntermediate));
    if (cudaStatus != hipSuccess) {
        return cudaStatus;
    }

    // Define grid and block dimensions
    dim3 blockDim(16, 16);
    dim3 gridDim((raysPerLine + blockDim.x - 1) / blockDim.x,
        (totalLines + blockDim.y - 1) / blockDim.y);

    // Launch kernel
    ProcessRaysKernel << <gridDim, blockDim >> > (*rayParams, *renderParams, raysPerLine, totalLines, d_results);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {      
        hipFree(d_results);
        return cudaStatus;
    }

    // Allocate host memory
    RayDataTypeIntermediate* h_results = new RayDataTypeIntermediate[raysPerLine * totalLines];

    // Copy results back to host
    cudaStatus = hipMemcpy(h_results, d_results, raysPerLine * totalLines * sizeof(RayDataTypeIntermediate), hipMemcpyDeviceToHost);

    if (cudaStatus != hipSuccess) {
        std::cout << "Cuda error:" << hipGetErrorString(cudaStatus) << std::endl;
        delete[] h_results;
        hipFree(d_results);
        return cudaStatus;
    }

    // Process results and call the callback for each ray
    for (int rowCount = 0; rowCount < totalLines; ++rowCount) {
        for (int rayCount = 0; rayCount < raysPerLine; ++rayCount) {
            int index = rowCount * raysPerLine + rayCount;
            if (callback) {
                callback(rayCount, rowCount, &h_results[index]);
            }
        }
    }

    // Free memory
    delete[] h_results;
    hipFree(d_results);

    return hipSuccess;
}