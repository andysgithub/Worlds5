#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>
#include <cmath>
#include <stdio.h>
#include "inline.cuh"
#include "cuda_interface.h"
#include "RayTracer.cuh"
#include "RayProcessing.cuh"

__constant__ RayTracingParams d_params;

// Host function to initialize the GPU with constant parameters
extern "C" hipError_t InitializeGPUKernel(const RayTracingParams* params)
{
    // Copy the parameters to the device's constant memory
    return hipMemcpyToSymbol((const void*)&d_params, (const void*)params, sizeof(RayTracingParams));

    void* d_addr;
    hipError_t error;

    // Get the address of the symbol in device memory
    error = hipGetSymbolAddress(&d_addr, HIP_SYMBOL((const void*))&d_params);
    if (error != hipSuccess) {
        return error;
    }

    // Copy the data to the symbol
    error = hipMemcpy(d_addr, params, sizeof(RayTracingParams), hipMemcpyHostToDevice);
    return error;
}

extern "C" hipError_t InitializeTransformMatrix(const float* positionMatrix)
{
    return hipMemcpyToSymbol(HIP_SYMBOL(cudaTrans), positionMatrix, sizeof(float) * DimTotal * (DimTotal + 1));
}

extern "C" void launchProcessRayKernel(RayTracingParams rayParams, RenderingParams renderParams,
    int raysPerLine, int totalLines)
{
    // Allocate device memory
    RayDataTypeIntermediate* d_results;
    hipMalloc(&d_results, raysPerLine * totalLines * sizeof(RayDataTypeIntermediate));

    // Define grid and block dimensions
    dim3 blockDim(16, 16);
    dim3 gridDim((raysPerLine + blockDim.x - 1) / blockDim.x,
        (totalLines + blockDim.y - 1) / blockDim.y);

    // Launch kernel
    ProcessRayKernel << <gridDim, blockDim >> > (rayParams, renderParams, raysPerLine, totalLines, d_results);

    // Allocate host memory and copy results back
    RayDataTypeIntermediate* h_results = new RayDataTypeIntermediate[raysPerLine * totalLines];
    hipMemcpy(h_results, d_results, raysPerLine * totalLines * sizeof(RayDataTypeIntermediate), hipMemcpyDeviceToHost);

    // Free memory
    delete[] h_results;
    hipFree(d_results);
}