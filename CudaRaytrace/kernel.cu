#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>
#include <cmath>
#include <stdio.h>
#include "inline.cuh"
#include "cuda_interface.h"
#include "RayTracer.cuh"

__constant__ RayTracingParams d_params;

// Host function to initialize the GPU with constant parameters
extern "C" hipError_t InitializeGPUKernel(const RayTracingParams* params)
{
    // Copy the parameters to the device's constant memory
    return hipMemcpyToSymbol((const void*)&d_params, (const void*)params, sizeof(RayTracingParams));

    void* d_addr;
    hipError_t error;

    // Get the address of the symbol in device memory
    error = hipGetSymbolAddress(&d_addr, HIP_SYMBOL((const void*))&d_params);
    if (error != hipSuccess) {
        return error;
    }

    // Copy the data to the symbol
    error = hipMemcpy(d_addr, params, sizeof(RayTracingParams), hipMemcpyHostToDevice);
    return error;
}

extern "C" hipError_t InitializeTransformMatrix(const float* positionMatrix)
{
    return hipMemcpyToSymbol(HIP_SYMBOL(cudaTrans), positionMatrix, sizeof(float) * DimTotal * (DimTotal + 1));
}

extern "C" int launchTraceRayKernel(float XFactor, float YFactor, float ZFactor, int rayPoints,
    int* externalPoints, float* modulusValues, float* angles, float* distances)
{
    // Allocate device memory
    int* d_externalPoints, * d_recordedPointsOut;
    float* d_modulusValues, * d_angles;
    float* d_distances;

    hipMalloc(&d_externalPoints, rayPoints * sizeof(int));
    hipMalloc(&d_modulusValues, rayPoints * sizeof(float));
    hipMalloc(&d_angles, rayPoints * sizeof(float));
    hipMalloc(&d_distances, rayPoints * sizeof(float));
    hipMalloc(&d_recordedPointsOut, sizeof(int));

    // Launch kernel
    TraceRayKernel<<<1, 1>>>(XFactor, YFactor, ZFactor, rayPoints,
        d_externalPoints, d_modulusValues, d_angles, d_distances, d_recordedPointsOut);

    // Copy results back to host
    int recordedPoints;
    hipMemcpy(externalPoints, d_externalPoints, rayPoints * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(modulusValues, d_modulusValues, rayPoints * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(angles, d_angles, rayPoints * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(distances, d_distances, rayPoints * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&recordedPoints, d_recordedPointsOut, sizeof(int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_externalPoints);
    hipFree(d_modulusValues);
    hipFree(d_angles);
    hipFree(d_distances);
    hipFree(d_recordedPointsOut);

    return recordedPoints;
}