#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>
#include <cmath>
#include <stdio.h>
#include "inline.cuh"
#include "cuda_interface.h"
#include "vectors.cuh"
#include "RayTracer.cuh"

#ifdef __INTELLISENSE__
#define __any_sync(x, y) (y)
#define __activemask() 0
#define HIP_INF_F 0
#endif

__constant__ float cudaTrans[6][DimTotal];
__constant__ RayTracingParams d_rayParams;
__constant__ RenderingParams d_renderParams;

namespace RayTracer {

    __device__ int TraceRay(float startDistance, Vector3 rayPoint, int rayPoints,
        int* __restrict__ externalPoints, float* __restrict__ modulusValues,
        float* __restrict__ angles, float* __restrict__ distances) {

        const vector5Single c = { 0, 0, 0, 0, 0 };
        float Modulus, Angle, currentDistance = startDistance;
        int recordedPoints = 0, sampleCount = 0;
        bool externalPoint = SamplePoint(currentDistance, &Modulus, &Angle, rayPoint, c);

        if (recordedPoints < rayPoints) {
            externalPoints[recordedPoints] = externalPoint ? 1 : 0;
            modulusValues[recordedPoints] = Modulus;
            angles[recordedPoints] = Angle;
            distances[recordedPoints] = currentDistance;
            recordedPoints++;
        }

        if (d_rayParams.activeIndex == 0) {
            bool previousPointExternal = true;

            float stepFactor = d_rayParams.surfaceSmoothing / 10;
            float stepSize = -d_rayParams.samplingInterval * stepFactor;

            while (recordedPoints < rayPoints && sampleCount < d_rayParams.maxSamples) {
                currentDistance += d_rayParams.samplingInterval;
                sampleCount++;

                externalPoint = SamplePoint(currentDistance, &Modulus, &Angle, rayPoint, c);

                bool shouldRecord = !externalPoint && previousPointExternal;

                if (shouldRecord) {
                    float sampleDistance = FindSurface(stepSize, stepFactor, currentDistance, rayPoint);
                    //float sampleDistance = currentDistance;

                    bool foundGap = gapFound(sampleDistance, rayPoint, c);

                    if (d_rayParams.surfaceThickness > 0 && foundGap) {
                        previousPointExternal = true;
                        continue;
                    }
                    externalPoint = SamplePoint(sampleDistance, &Modulus, &Angle, rayPoint, c);

                    externalPoints[recordedPoints] = externalPoint ? 1 : 0;
                    modulusValues[recordedPoints] = Modulus;
                    angles[recordedPoints] = Angle;
                    distances[recordedPoints] = sampleDistance;
                    recordedPoints++;
                }

                previousPointExternal = externalPoint;
            }
        }

        if (d_rayParams.activeIndex == 1) {
            while (recordedPoints < rayPoints && sampleCount < d_rayParams.maxSamples) {
                currentDistance += d_rayParams.samplingInterval;
                sampleCount++;

                externalPoint = SamplePoint(currentDistance, &Modulus, &Angle, rayPoint, c);

                ///// Set value for external point /////

                float angleChange = fabs(Angle - angles[recordedPoints - 1]);

                // If orbit value is sufficiently different from the last recorded sample
                if (angleChange > d_rayParams.boundaryInterval) {
                    // Perform binary search between this and the recorded point, to determine boundary position
                    float sampleDistance = FindBoundary(currentDistance, angles[recordedPoints - 1],
                        &externalPoint, &Modulus, &Angle, rayPoint);

                    // Save this point value in the ray collection
                    externalPoints[recordedPoints] = externalPoint ? 1 : 0;
                    modulusValues[recordedPoints] = Modulus;
                    angles[recordedPoints] = Angle;
                    distances[recordedPoints] = sampleDistance;
                    recordedPoints++;
                }
            }
        }
        
        if (recordedPoints < rayPoints) {
            distances[recordedPoints] = HIP_INF_F;
        }
        return recordedPoints + 1;
    }

    // Perform a binary search to refine the surface position
    __device__ float FindSurface(
        float stepSize, float stepFactor, float currentDistance,
        Vector3 rayPoint) {

        float sampleDistance = currentDistance;
        const vector5Single c = { 0, 0, 0, 0, 0 };

        for (int i = 0; i < d_rayParams.binarySearchSteps; i++) {
            sampleDistance += stepSize;
            stepSize = fabs(stepSize) * stepFactor;

            bool isExternal = SamplePoint(sampleDistance, rayPoint, c);

            // If inside the fractal, step back next time
            if (!isExternal) {
                stepSize = -stepSize;
            }
        }
        return sampleDistance;
    }

    __device__ float FindBoundary(float currentDistance, float previousAngle,
        bool* externalPoint, float* Modulus, float* Angle,
        Vector3 rayPoint) {

        float stepSize = -d_rayParams.samplingInterval / 2;
        float sampleDistance = currentDistance;
        const vector5Single c = { 0, 0, 0, 0, 0 };

        #pragma unroll 1
        for (int i = 0; i < d_rayParams.binarySearchSteps; i++) {
            sampleDistance += stepSize;
            *externalPoint = SamplePoint(sampleDistance, Modulus, Angle, rayPoint, c);

            float angleChange = fabsf(*Angle - previousAngle);
            bool exceedsBoundary = (angleChange > d_rayParams.boundaryInterval);

            // Use a branchless approach to update stepSize
            float stepSizeAbs = fabsf(stepSize);
            stepSize = copysignf(stepSizeAbs / 2, exceedsBoundary ? -stepSizeAbs : stepSizeAbs);
        }

        return sampleDistance;
    }

    __device__ bool SamplePoint(float distance, float* Modulus, float* Angle, Vector3 rayPoint, vector5Single c) {
        // Determine the x,y,z coord for this point
        Vector3 imagePoint = Vector3(distance * rayPoint.X, distance * rayPoint.Y, distance * rayPoint.Z);

        // Transform 3D point x,y,z into nD fractal space at point c[]
        VectorTrans(imagePoint, &c);

        constexpr float PI = 3.1415926536f;
        constexpr int MaxCount = 100;
        vector5Single z = { 0 };
        vector5Single diff;
        float ModulusTotal = 0;
        float AngleTotal = PI;

        z.coords[DimTotal - 2] = 0;
        z.coords[DimTotal - 1] = 0;
        v_mov(c.coords, z.coords);

        vector5Single vectorSet[3];
        v_mov(z.coords, vectorSet[1].coords);

        float bailout_squared = d_rayParams.bailout * d_rayParams.bailout;
        int count = 0;
        bool escaped = false;

        #pragma unroll 1
        for (; count < MaxCount; count++) {
            v_mandel(z.coords, c.coords);
            v_mov(z.coords, vectorSet[2].coords);

            if (count > 0 && count < 10) {
                AngleTotal += vectorAngle(vectorSet[0], vectorSet[1], vectorSet[2]);
            }

            v_subm(c.coords, z.coords, diff.coords);
            float ModVal = v_mod(diff.coords);
            ModulusTotal += ModVal;

            escaped = (ModVal > bailout_squared);
            if (__any_sync(__activemask(), escaped)) {
                if (!escaped) count = MaxCount;
                break;
            }

            v_mov(vectorSet[1].coords, vectorSet[0].coords);
            v_mov(vectorSet[2].coords, vectorSet[1].coords);
        }

        *Modulus = ModulusTotal / (count + 1);
        *Angle = AngleTotal / (count < 10 ? count + 1 : 10);

        return escaped;
    }

    __device__ bool SamplePoint(float distance, Vector3 rayPoint, vector5Single c) {
        // Determine the x,y,z coord for this point
        Vector3 imagePoint = Vector3(distance * rayPoint.X, distance * rayPoint.Y, distance * rayPoint.Z);

        // Transform 3D point x,y,z into nD fractal space at point c[]
        VectorTrans(imagePoint, &c);

        // Determine orbit value for this point
        constexpr int MaxCount = 1000;  // Use int instead of long for better performance on GPUs
        vector5Single z = { 0 };
        vector5Single diff;
        float ModulusTotal = 0;
        float bailout_squared = d_rayParams.bailout * d_rayParams.bailout;

        z.coords[DimTotal - 2] = 0;
        z.coords[DimTotal - 1] = 0;
        v_mov(c.coords, z.coords);

        #pragma unroll 1
        for (int count = 0; count < MaxCount; ++count) {
            v_mandel(z.coords, c.coords);  // z = z*z + c

            // Determine modulus for this point in orbit
            v_subm(c.coords, z.coords, diff.coords);  // Current orbit size = mod(z - c)
            float ModVal = v_mod(diff.coords);

            // Check if point has escaped
            if (ModVal > bailout_squared) return true;

            // Accumulate modulus value
            ModulusTotal += ModVal;
        }

        return false;
    }

    __device__ bool gapFound(float currentDistance, Vector3 rayPoint, vector5Single c) {
        float testDistance;

        #pragma unroll
        for (int factor = 1; factor <= 4; factor++) {
            testDistance = currentDistance + d_rayParams.surfaceThickness * factor / 4;
            if (SamplePoint(testDistance, rayPoint, c)) {
                return true;
            }
        }

        return false;
    }

    __device__ void VectorTrans(Vector3 imagePoint, vector5Single* c) {
        for (int col = 0; col < DimTotal; col++) {
            (*c).coords[col] =
                cudaTrans[0][col] * imagePoint.X +
                cudaTrans[1][col] * imagePoint.Y +
                cudaTrans[2][col] * imagePoint.Z +
                cudaTrans[5][col];
        }
    }

    __device__ float vectorAngle(const vector5Single& A, const vector5Single& B, const vector5Single& C) {
        float v1[5], v2[5];
        float dot1 = 0.0f, dot2 = 0.0f, dotProduct = 0.0f;

        #pragma unroll
        for (int i = 0; i < 5; ++i) {
            v1[i] = B.coords[i] - A.coords[i];
            v2[i] = B.coords[i] - C.coords[i];
            dot1 += v1[i] * v1[i];
            dot2 += v2[i] * v2[i];
        }

        // Use a small epsilon value to avoid division by zero
        const float epsilon = 1e-6f;
        float invMod1 = 1.0f / sqrtf(fmaxf(dot1, epsilon));
        float invMod2 = 1.0f / sqrtf(fmaxf(dot2, epsilon));

        #pragma unroll
        for (int i = 0; i < 5; ++i) {
            dotProduct += (v1[i] * invMod1) * (v2[i] * invMod2);
        }

        // Clamp dotProduct to [-1, 1] range
        dotProduct = fmaxf(-1.0f, fminf(1.0f, dotProduct));
        return acosf(dotProduct);
    }
}