#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>
#include <cmath>
#include <stdio.h>
#include "inline.cuh"
#include "cuda_interface.h"
#include "RayTracer.cuh"

__constant__ float cudaTrans[6][DimTotal];

namespace RayTracer {
    __device__ void VectorTrans2(float x, float y, float z, vector5Single* c) {
        for (int col = 0; col < DimTotal; col++) {
            (*c).coords[col] =
                cudaTrans[0][col] * x +
                cudaTrans[1][col] * y +
                cudaTrans[2][col] * z +
                cudaTrans[5][col];
        }
    }

    __device__ float vectorAngle(const vector5Single& A, const vector5Single& B, const vector5Single& C) {
        vector5Single v1, v2;
        float dotProduct = 0.0;

        // Vector v1 = B - A 
        v1.coords[0] = B.coords[0] - A.coords[0];
        v1.coords[1] = B.coords[1] - A.coords[1];
        v1.coords[2] = B.coords[2] - A.coords[2];
        v1.coords[3] = B.coords[3] - A.coords[3];
        v1.coords[4] = B.coords[4] - A.coords[4];

        float modulus = sqrt(v1.coords[0] * v1.coords[0] + v1.coords[1] * v1.coords[1] +
            v1.coords[2] * v1.coords[2] + v1.coords[3] * v1.coords[3] +
            v1.coords[4] * v1.coords[4]);

        if (modulus != 0.0) {
            float factor = 1.0 / modulus;

            // Normalize v1 by dividing by mod(v1)
            v1.coords[0] *= factor;
            v1.coords[1] *= factor;
            v1.coords[2] *= factor;
            v1.coords[3] *= factor;
            v1.coords[4] *= factor;

            // Vector v2 = B - C 
            v2.coords[0] = B.coords[0] - C.coords[0];
            v2.coords[1] = B.coords[1] - C.coords[1];
            v2.coords[2] = B.coords[2] - C.coords[2];
            v2.coords[3] = B.coords[3] - C.coords[3];
            v2.coords[4] = B.coords[4] - C.coords[4];

            modulus = sqrt(v2.coords[0] * v2.coords[0] + v2.coords[1] * v2.coords[1] +
                v2.coords[2] * v2.coords[2] + v2.coords[3] * v2.coords[3] +
                v2.coords[4] * v2.coords[4]);

            if (modulus != 0.0) {
                factor = 1.0 / modulus;

                // Normalize v2 by dividing by mod(v2)
                v2.coords[0] *= factor;
                v2.coords[1] *= factor;
                v2.coords[2] *= factor;
                v2.coords[3] *= factor;
                v2.coords[4] *= factor;

                // Calculate dot product of v1 and v2
                dotProduct = v1.coords[0] * v2.coords[0] + v1.coords[1] * v2.coords[1] +
                    v1.coords[2] * v2.coords[2] + v1.coords[3] * v2.coords[3] +
                    v1.coords[4] * v2.coords[4];
            }
        }

        // Clamp dotProduct to the range [-1, 1]
        dotProduct = fmaxf(fminf(dotProduct, 1.0f), -1.0f);

        // Return the angle in radians
        return acos(dotProduct);
    }

    // Determine whether nD point c[] in within the set
    // Returns true if point is external to the set
    __device__ bool ExternalPoint2(vector5Single c, float bailout)
    {
        const long MaxCount = (long)(1000);		        // Iteration count for external points
        vector5Single z;										// Temporary 5-D vector
        vector5Single diff;										// Temporary 5-D vector for orbit size
        float ModulusTotal = 0;
        float ModVal = 0;
        long count;

        z.coords[DimTotal - 2] = 0;
        z.coords[DimTotal - 1] = 0;

        v_mov(c.coords, z.coords);        // z = c

        for (count = 0; count < MaxCount; count++)
        {
            v_mandel(z.coords, c.coords);                   //    z = z*z + c

            // Determine modulus for this point in orbit
            v_subm(c.coords, z.coords, diff.coords);        // Current orbit size = mod(z - c)
            ModVal = v_mod(diff.coords);

            // Accumulate modulus value
            ModulusTotal += ModVal;

            // Stop accumulating values when modulus exceeds bailout value
            if (ModVal > bailout * bailout)
            {
                count++;
                break;
            }
        }

        // Return true if this point is external to the set
        return (count < MaxCount);
    }

    __device__ bool ProcessPoint2(float* Modulus, float* Angle, float bailout, vector5Single c) {
        float const PI = 3.1415926536;

        const long MaxCount = (long)100;
        vector5Single z;
        vector5Single diff;
        float ModulusTotal = 0;
        float ModVal = 0;
        float AngleTotal = PI;
        long count;

        z.coords[DimTotal - 2] = 0;
        z.coords[DimTotal - 1] = 0;

        v_mov(c.coords, z.coords);
        vector5Single vectorSet[3];
        v_mov(z.coords, vectorSet[1].coords);

        for (count = 0; count < MaxCount; count++) {
            v_mandel(z.coords, c.coords);
            v_mov(z.coords, vectorSet[2].coords);

            if (count > 0 && count < 10) {
                AngleTotal += vectorAngle(vectorSet[0], vectorSet[1], vectorSet[2]);
            }

            v_subm(c.coords, z.coords, diff.coords);
            ModVal = v_mod(diff.coords);

            ModulusTotal += ModVal;

            if (ModVal > bailout * bailout) {
                count++;
                break;
            }

            v_mov(vectorSet[1].coords, vectorSet[0].coords);
            v_mov(vectorSet[2].coords, vectorSet[1].coords);
        }

        *Modulus = (float)(ModulusTotal / count);
        *Angle = (float)(AngleTotal / (count > 10 ? 10 : count + 1));

        return (count < MaxCount);
    }

    __device__ bool SamplePoint2(float distance, float* Modulus, float* Angle, float bailout, float xFactor, float yFactor, float zFactor, vector5Single c) {
        // Determine the x,y,z coord for this point
        const float XPos = distance * xFactor;
        const float YPos = distance * yFactor;
        const float ZPos = distance * zFactor;

        // Transform 3D point x,y,z into nD fractal space at point c[]
        VectorTrans2(XPos, YPos, ZPos, &c);

        // Determine orbit value for this point
        return ProcessPoint2(Modulus, Angle, bailout, c);
    }

    __device__ bool SamplePoint2(float distance, float bailout, float xFactor, float yFactor, float zFactor, vector5Single c) {
        // Determine the x,y,z coord for this point
        const float XPos = distance * xFactor;
        const float YPos = distance * yFactor;
        const float ZPos = distance * zFactor;

        // Transform 3D point x,y,z into nD fractal space at point c[]
        VectorTrans2(XPos, YPos, ZPos, &c);

        // Determine orbit value for this point
        return ExternalPoint2(c, bailout);
    }

    __device__ bool gapFound2(float currentDistance, float surfaceThickness, float xFactor, float yFactor, float zFactor, float bailout, vector5Single c) {
        float testDistance;

        for (int factor = 1; factor <= 4; factor++) {
            testDistance = currentDistance + surfaceThickness * factor / 4;

            if (SamplePoint2(testDistance, bailout, xFactor, yFactor, zFactor, c)) {
                return true;
            }
        }
        return false;
    }

    __device__ float FindSurface2(
        float samplingInterval, float surfaceSmoothing, int binarySearchSteps, float currentDistance,
        float xFactor, float yFactor, float zFactor, float bailout) {

        float stepFactor = surfaceSmoothing / 10;
        float stepSize = -samplingInterval * stepFactor;
        float sampleDistance = currentDistance;
        const vector5Single c = { 0, 0, 0, 0, 0 };

        for (int i = 0; i < binarySearchSteps; i++) {
            sampleDistance += stepSize;

            if (!SamplePoint2(sampleDistance, bailout, xFactor, yFactor, zFactor, c)) {
                stepSize = -fabs(stepSize) * stepFactor;
            }
            else {
                stepSize = fabs(stepSize) * stepFactor;
            }
        }
        return sampleDistance;
    }

    __device__ float FindBoundary2(float samplingInterval, int binarySearchSteps, float currentDistance, float previousAngle,
        float boundaryInterval, bool* externalPoint, float* Modulus, float* Angle,
        float xFactor, float yFactor, float zFactor, float bailout) {
        float stepSize = -samplingInterval / 2;
        float sampleDistance = currentDistance;
        const vector5Single c = { 0, 0, 0, 0, 0 };

        for (int i = 0; i < binarySearchSteps; i++) {
            sampleDistance += stepSize;
            *externalPoint = SamplePoint2(sampleDistance, Modulus, Angle, bailout, xFactor, yFactor, zFactor, c);

            const float angleChange = fabs(*Angle - previousAngle);

            if (angleChange > boundaryInterval) {
                stepSize = -fabs(stepSize) / 2;
            }
            else {
                stepSize = fabs(stepSize) / 2;
            }
        }
        return sampleDistance;
    }
}