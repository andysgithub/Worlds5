#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include ""
#include <vector>
#include <cmath>
#include <iostream>
#include <stdio.h>
#include "cuda_interface.h"
#include "Vectors.cuh"
#include "RayTracer.cuh"
#include "RayProcessing.cuh"
//#include "Clipping.h" 

__device__ void FillIntermediateResult(
    int* externalPoints, float* modulusValues, float* angleValues, float* distanceValues, 
    int points, int maxPoints, RayDataTypeIntermediate* result)
{
    result->ArraySize = (points < maxPoints) ? points : maxPoints;
    result->BoundaryTotal = points;

    for (int i = 0; i < result->ArraySize; ++i) {
        result->ExternalPoints[i] = externalPoints[i];
        result->ModulusValues[i] = modulusValues[i];
        result->AngleValues[i] = angleValues[i];
        result->DistanceValues[i] = distanceValues[i];
    }
}

__device__ RayDataTypeIntermediate ConvertToIntermediate(const RayDataType& original, int maxSize) {
    RayDataTypeIntermediate result;
    result.ArraySize = original.ArraySize;
    result.BoundaryTotal = original.BoundaryTotal;

    // Assuming RayDataTypeIntermediate has been modified to use fixed-size arrays
    for (int i = 0; i < result.ArraySize && i < maxSize; ++i) {
        result.ExternalPoints[i] = original.ExternalPoints[i];
        result.ModulusValues[i] = original.ModulusValues[i];
        result.AngleValues[i] = original.AngleValues[i];
        result.DistanceValues[i] = original.DistanceValues[i];
    }

    return result;
}

__device__ void ProcessRayKernel(int rayCountX, int rayCountY, RayDataTypeIntermediate* result)
{
    float latitude = d_rayParams.latitudeStart - rayCountY * d_rayParams.angularResolution;
    float longitude = d_rayParams.longitudeStart - rayCountX * d_rayParams.angularResolution;

    float latRadians = latitude * DEG_TO_RAD;
    float longRadians = longitude * DEG_TO_RAD;

    Vector3 rayPoint = Vector3(
        cosf(latRadians) * sinf(-longRadians),
        sinf(latRadians),
        cosf(latRadians) * cosf(-longRadians));

    //float xFactor = cosf(latRadians) * sinf(-longRadians);
    //float yFactor = sinf(latRadians);
    //float zFactor = cosf(latRadians) * cosf(-longRadians);

    float startDistance = d_rayParams.sphereRadius;

    //if (rayParams.useClipping) {
    //    float distance = CalculateDistance(latRadians, longRadians, d_rayParams.clippingAxes, d_rayParams.clippingOffset);
    //    if (distance > startDistance) startDistance = distance;
    //}

    int externalPoints[MAX_POINTS];
    float modulusValues[MAX_POINTS];
    float angleValues[MAX_POINTS];
    float distanceValues[MAX_POINTS];

    int points = RayTracer::TraceRay(startDistance,
        rayPoint, (int)MAX_POINTS,
        externalPoints, modulusValues, angleValues, distanceValues);

    // Directly fill the intermediate result
    FillIntermediateResult(externalPoints, modulusValues, angleValues, distanceValues, points, MAX_POINTS, result);
}

// CUDA kernel function
__global__ void ProcessRaysKernel(int raysPerLine, int totalLines, RayDataTypeIntermediate* results)
{
    int rayCountX = blockIdx.x * blockDim.x + threadIdx.x;
    int rayCountY = blockIdx.y * blockDim.y + threadIdx.y;

    if (rayCountX >= raysPerLine || rayCountY >= totalLines)
        return;

    int index = rayCountY * raysPerLine + rayCountX;
    ProcessRayKernel(rayCountX, rayCountY, &results[index]);

    //printf("Ray %d processed\n", index);
}
